#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <stdio.h>
#include "sar_bp.hpp"

__global__ 
void GPU_unit_test_device_fcns( const float3 * a_signal,
   			               float3 * a_interp_grid,
             			 const int      a_length_signal,
			         const int      a_length_interp, 
			         const float    a_conv_factor,
			         const float    a_offset)


{

  // Run first device function, linear_interpolation

  int i_loop_max = a_length_interp / a_length_signal;                                                                                             
                                                                                                                                                                     
  for(int i_loop = 0; i_loop < i_loop_max; i_loop++)                                                                                                                 
    { 
      int i_interp = threadIdx.x + i_loop * a_length_signal; 
      linear_interpolation(a_signal,
		       a_interp_grid,
		       a_length_signal,
		       a_length_interp,
		       a_conv_factor,
		       a_offset,
		       threadIdx.x,
		       i_interp);
		       
    }

}

/*
  * unit_test_linear_interp
  * 
  * This is a unit test of the linear interpolation function
  * It fills an signal and interpolation grid buffer with data
  * and runs the linear interpolation function. Data selected covers 
  * both buffer walk-off edge cases, positive and negative slopes, and 
  * positive and negative values
  */
__host__
void CPU_unit_test_device_fcns()
{

  // Specify a signal to interpolate.
  // For ease of testing, only use three points
  size_t signal_size = 3;
  float3 signal[signal_size];
  signal[0] = make_float3(0.0, 5.0, 0.0);
  signal[1] = make_float3(1.0, 10.0, 0.0);
  signal[2] = make_float3(2.0, 6.0, 0.0);

  // Specify a grid of positions to interpolate on
  size_t interp_grid_size = 9;
  float3 interp_grid[interp_grid_size];
  interp_grid[0] = make_float3(-1.0, 0.0, 0.0);
  interp_grid[1] = make_float3(-0.5, 0.0, 0.0);
  interp_grid[2] = make_float3(0.0, 0.0, 0.0);
  interp_grid[3] = make_float3(0.5, 0.0, 0.0);
  interp_grid[4] = make_float3(1.0, 0.0, 0.0);
  interp_grid[5] = make_float3(1.5, 0.0, 0.0);
  interp_grid[6] = make_float3(2.0, 0.0, 0.0);
  interp_grid[7] = make_float3(2.5, 0.0, 0.0);
  interp_grid[8] = make_float3(3.0, 0.0, 0.0);


  for(int ii = 0; ii < interp_grid_size; ii++)
    {
      std::cout << "On CPU interp_grid.x" << interp_grid[ii].x << ".y: " << interp_grid[ii].y << std::endl;
    } 

  // Conversion factor is how many signal points per interpolation point
  // This is used to select which signal point the linear interpolator uses for a
  // interpolation grid point

  float conv_factor = float(signal_size) / float(interp_grid_size);
  float offset = interp_grid[0].x - signal[0].x;

  // Create device buffers for the data
  float3* d_signal;
  float3* d_interp_grid;
  // Allocate and copy over the signal and interpolation grid
  
  hipMalloc(&d_signal, signal_size*sizeof(float3));
  hipMalloc(&d_interp_grid, interp_grid_size*sizeof(float3));

  hipMemcpy(d_signal,
	     &signal,
	     signal_size*sizeof(float3),
	     hipMemcpyHostToDevice); 

  hipMemcpy(d_interp_grid,
	     &interp_grid,
	     interp_grid_size*sizeof(float3),
	     hipMemcpyHostToDevice); 

  // Launch kernel
  GPU_unit_test_device_fcns<<<1, 128>>>( d_signal,
					 d_interp_grid,
					 signal_size,
					 interp_grid_size,
					 conv_factor,
					 offset);

  // Copy back data, and print results
  hipMemcpy(interp_grid,
	     d_interp_grid,
	     interp_grid_size*sizeof(float3),
	     hipMemcpyDeviceToHost);

  std::cout << "Results:\n";
  for(int ii = 0; ii < interp_grid_size; ii++)
    {
      std::cout << "x: " << interp_grid[ii].x
		<< "  y: " << interp_grid[ii].y << "\n";
    }

  // cleanup
  hipFree( d_signal );
  hipFree( d_interp_grid);

}


int main(int argc, char** argv)
{
  CPU_unit_test_device_fcns();
}

